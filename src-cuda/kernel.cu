#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <fstream>
#include <string>
#include <iostream>
#include <sstream>
#include <cstring>
#include <cstdio>
#include <math.h>
#include <list>
#include "distance_matrix.h"
#include "algorithm_engineering.h"
#include "edge_dali.h"
#include "LR.h"

using namespace std;

int nb_row;
int nb_col;
int nb_branch_node;
int nb_solved_node(0);
double ** row_mat;
double ** col_mat;
double **** align_edge;
double global_ub(INFINITY);
double global_lb(-INFINITY);
int * alignment = new int[200];
string filename1;
string filename2;

list<problem> problem_list;

void insert_node(problem);
void delete_node();
void end();

int main()
{
	struct timeb startTime, endTime;

	int nb_iteration;
	int precise;
	double dali_score;
	double z_score;

	cout << "nb_B&B_node:	";
	cin >> nb_branch_node;
	cout << "nb_iteration:	";
	cin >> nb_iteration;
	cout << "protein  1:	";
	cin >> filename1;
	cout << "protein  2:	";
	cin >> filename2;
	cout << "Z score:	";
	cin >> z_score;
	cout << "precise:	";
	cin >> precise;

	ftime(&startTime);

	row_mat = distanceMatrix(filename1, &nb_row);
	col_mat = distanceMatrix(filename2, &nb_col);

	double L = sqrt((double)nb_row * (double)nb_col);
	double x = (L > 400.0) ? 400.0 : L;
	double m_L = 7.9494 + 0.70852*x + 2.5895*0.0001*x*x - 1.9156*0.000001*x*x*x;
	if (L > 400.0)
	{
		m_L = m_L + (L - 400.0)*1.0;
	}
	dali_score = z_score * 0.5 * m_L + m_L;
	cout << "dali score:	" << dali_score << endl;

	int ** domain;
	int align_edge_size = align_edge_size_compute(nb_row, nb_col);
	double * gpu_align_edge;
	hipMalloc((void**)&gpu_align_edge, sizeof(double) * align_edge_size);

	double * gpu_row_mat;
	double * gpu_col_mat;
	hipMalloc((void**)&gpu_row_mat, sizeof(double) * nb_row * nb_row);
	hipMalloc((void**)&gpu_col_mat, sizeof(double) * nb_col * nb_col);
	double * cpu_row_mat = new double[nb_row * nb_row];
	double * cpu_col_mat = new double[nb_col * nb_col];
	for (int i = 0; i < nb_row; i++)
	{
		for (int j = 0; j < nb_row; j++)
			cpu_row_mat[i * nb_row + j] = row_mat[i][j];
	}
	for (int i = 0; i < nb_col; i++)
	{
		for (int j = 0; j < nb_col; j++)
			cpu_col_mat[i * nb_col + j] = col_mat[i][j];
	}
	hipMemcpy(gpu_row_mat, cpu_row_mat, sizeof(double) * nb_row * nb_row, hipMemcpyHostToDevice);
	hipMemcpy(gpu_col_mat, cpu_col_mat, sizeof(double) * nb_col * nb_col, hipMemcpyHostToDevice);

	gpu_edge_dalix << <nb_row * nb_col / 64 + 1, 64 >> >(gpu_align_edge, gpu_row_mat, gpu_col_mat, nb_row, nb_col);

	double * cpu_align_edge = new double[align_edge_size];
	hipMemcpy(cpu_align_edge, gpu_align_edge, sizeof(double) * align_edge_size, hipMemcpyDeviceToHost);

	domain = algorithm_engineering(nb_row, nb_col, dali_score, gpu_align_edge); //algorithm engineering

	int * up = new int[nb_col];
	int * lo = new int[nb_col];
	for (int col = 0; col < nb_col; col++)
	{
		lo[col] = -1;
		up[col] = nb_row;
		for (int row = nb_row - 1; domain[row][col] == 0 && row >= 0; row--)
			up[col] = row;
		for (int row = 0; domain[row][col] == 0 && row < nb_row; row++)
			lo[col] = row;
	}

	int *** lambda_for_cik = new int **[nb_row];
	for (int row = 0; row < nb_row; row++)
	{
		lambda_for_cik[row] = new int *[nb_col];
		for (int col = 0; col < nb_col; col++)
		{
			if (row < up[col] && row > lo[col])
			{
				lambda_for_cik[row][col] = new int[(nb_row - row - 1)*(nb_col - col - 1) * 2 + 1];
				int i = 1;
				int nb_lessthan0 = 0;
				for (int row2 = 0; row2 < nb_row - row - 1; row2++)
				{
					for (int col2 = 0; col2 < nb_col - col - 1; col2++)
					{
						if (cpu_align_edge[align_edge_iter_compute(nb_row, nb_col, row, col, row2, col2)] <= 0)
						{
							lambda_for_cik[row][col][i++] = row + row2 + 1;
							lambda_for_cik[row][col][i++] = col + col2 + 1;
							nb_lessthan0++;
						}
					}
				}
				lambda_for_cik[row][col][0] = nb_lessthan0;
			}
		}
	}

	int cpu_lambda_for_cik_size(0);
	for (int row = 0; row < nb_row; row++)
	{
		for (int col = 0; col < nb_col; col++)
		{
			if (row < up[col] && row > lo[col])
			{
				cpu_lambda_for_cik_size += lambda_for_cik[row][col][0] * 2;
			}
		}
	}

	int * cpu_lambda_for_cik = new int[cpu_lambda_for_cik_size + nb_row * nb_col];
	int temp_iter(nb_row * nb_col);
	for (int row = 0; row < nb_row; row++)
	{
		for (int col = 0; col < nb_col; col++)
		{
			if (row < up[col] && row > lo[col])
			{
				cpu_lambda_for_cik[row * nb_col + col] = temp_iter;
				temp_iter += lambda_for_cik[row][col][0] * 2;
				for (int i = 0; i < lambda_for_cik[row][col][0] * 2; i++)
				{
					cpu_lambda_for_cik[cpu_lambda_for_cik[row * nb_col + col] + i] = lambda_for_cik[row][col][i + 1];
				}
			}
		}
	}

	problem root(INFINITY, -INFINITY, up, lo, nb_col);

	ftime(&endTime);
	cout << "Allocation time: " << (endTime.time - startTime.time) * 1000 + (endTime.millitm - startTime.millitm) << endl;
	ftime(&startTime);

	//-------------------------------------------------------------------------------root problem
	if (solve_lr(nb_row, nb_col, gpu_align_edge, cpu_lambda_for_cik, &root, cpu_lambda_for_cik_size, nb_iteration, alignment, precise, global_lb) == 0)
	{
		global_ub = root.ub;
		global_lb = root.lb;
		problem_list.push_front(root);
	}
	else
	{
		ftime(&endTime);
		cout << "Solve time: " << (endTime.time - startTime.time) * 1000 + (endTime.millitm - startTime.millitm) << endl;
		end();
		return 0;
	}
	//-------------------------------------------------------------------------------root problem

	//-------------------------------------------------------------------------------B & B
	int * lo1 = new int[nb_col];
	int * up1 = new int[nb_col];
	int * lo2 = new int[nb_col];
	int * up2 = new int[nb_col];

	while (!problem_list.empty() && nb_branch_node != 0)
	{
		problem current_problem(problem_list.front());
		problem_list.pop_front();
		split(nb_row, nb_col, current_problem, lo1, up1, lo2, up2);

		problem sub_problema(INFINITY, -INFINITY, up1, lo1, nb_col);
		problem sub_problemb(INFINITY, -INFINITY, up2, lo2, nb_col);
		split(nb_row, nb_col, sub_problema, lo1, up1, lo2, up2);
		problem sub_problem1(INFINITY, -INFINITY, up1, lo1, nb_col);
		problem sub_problem2(INFINITY, -INFINITY, up2, lo2, nb_col);
		split(nb_row, nb_col, sub_problemb, lo1, up1, lo2, up2);
		problem sub_problem3(INFINITY, -INFINITY, up1, lo1, nb_col);
		problem sub_problem4(INFINITY, -INFINITY, up2, lo2, nb_col);

		int return_value;

		return_value = solve_lr(nb_row, nb_col, gpu_align_edge, cpu_lambda_for_cik, &sub_problem1, cpu_lambda_for_cik_size, nb_iteration, alignment, precise, global_lb);
		if (return_value == 0)
		{
			if (sub_problem1.lb > global_lb)
			{
				global_lb = sub_problem1.lb;
				global_ub = sub_problem1.ub;
			}
			insert_node(sub_problem1);
		}
		else if (return_value == 2)
		{
			cout << "Useless node." << endl;
		}
		else
		{
			ftime(&endTime);
			cout << "Solve time: " << (endTime.time - startTime.time) * 1000 + (endTime.millitm - startTime.millitm) << endl;
			end();
			return 0;
		}

		nb_solved_node += 1;
		if (nb_solved_node >= nb_branch_node)
		{
			ftime(&endTime);
			cout << "Solve time: " << (endTime.time - startTime.time) * 1000 + (endTime.millitm - startTime.millitm) << endl;
			end();
			return 0;
		}

		return_value = solve_lr(nb_row, nb_col, gpu_align_edge, cpu_lambda_for_cik, &sub_problem2, cpu_lambda_for_cik_size, nb_iteration, alignment, precise, global_lb);
		if (return_value == 0)
		{
			if (sub_problem2.lb > global_lb)
			{
				global_lb = sub_problem2.lb;
				global_ub = sub_problem2.ub;
			}
			insert_node(sub_problem2);
		}
		else if (return_value == 2)
		{
			cout << "Useless node." << endl;
		}
		else
		{
			ftime(&endTime);
			cout << "Solve time: " << (endTime.time - startTime.time) * 1000 + (endTime.millitm - startTime.millitm) << endl;
			end();
			return 0;
		}

		nb_solved_node += 1;
		if (nb_solved_node >= nb_branch_node)
		{
			ftime(&endTime);
			cout << "Solve time: " << (endTime.time - startTime.time) * 1000 + (endTime.millitm - startTime.millitm) << endl;
			end();
			return 0;
		}

		return_value = solve_lr(nb_row, nb_col, gpu_align_edge, cpu_lambda_for_cik, &sub_problem3, cpu_lambda_for_cik_size, nb_iteration, alignment, precise, global_lb);
		if (return_value == 0)
		{
			if (sub_problem3.lb > global_lb)
			{
				global_lb = sub_problem3.lb;
				global_ub = sub_problem3.ub;
			}
			insert_node(sub_problem3);
		}
		else if (return_value == 2)
		{
			cout << "Useless node." << endl;
		}
		else
		{
			ftime(&endTime);
			cout << "Solve time: " << (endTime.time - startTime.time) * 1000 + (endTime.millitm - startTime.millitm) << endl;
			end();
			return 0;
		}

		nb_solved_node += 1;
		if (nb_solved_node >= nb_branch_node)
		{
			ftime(&endTime);
			cout << "Solve time: " << (endTime.time - startTime.time) * 1000 + (endTime.millitm - startTime.millitm) << endl;
			end();
			return 0;
		}

		return_value = solve_lr(nb_row, nb_col, gpu_align_edge, cpu_lambda_for_cik, &sub_problem4, cpu_lambda_for_cik_size, nb_iteration, alignment, precise, global_lb);
		if (return_value == 0)
		{
			if (sub_problem4.lb > global_lb)
			{
				global_lb = sub_problem4.lb;
				global_ub = sub_problem4.ub;
			}
			insert_node(sub_problem4);
		}
		else if (return_value == 2)
		{
			cout << "Useless node." << endl;
		}
		else
		{
			ftime(&endTime);
			cout << "Solve time: " << (endTime.time - startTime.time) * 1000 + (endTime.millitm - startTime.millitm) << endl;
			end();
			return 0;
		}

		nb_solved_node += 1;
		if (nb_solved_node >= nb_branch_node)
		{
			ftime(&endTime);
			cout << "Solve time: " << (endTime.time - startTime.time) * 1000 + (endTime.millitm - startTime.millitm) << endl;
			end();
			return 0;
		}

		delete_node();

		while (problem_list.size() > 20)
		{
			problem_list.pop_back();
		}
	}

	delete[] lo1, up1, lo2, up2;

	for (int row = 0; row < nb_row; row++)
	{
		for (int col = 0; col < nb_col; col++)
		{
			if (row < up[col] && row > lo[col])
			{
				delete[] lambda_for_cik[row][col];
			}
		}
		delete[] lambda_for_cik[row];
	}
	delete[] lambda_for_cik;
	delete[] up, lo;

	for (int row = 0; row < nb_row; row++)
	{
		delete[] domain[row];
	}
	delete[] domain;
	//-------------------------------------------------------------------------------B & B

	ftime(&endTime);
	cout << "Solve time: " << (endTime.time - startTime.time) * 1000 + (endTime.millitm - startTime.millitm) << endl;
	end();
	return 0;
}

void insert_node(problem current_problem)
{
	if (current_problem.ub >= global_lb)
	{
		int inserted(0);
		double new_lb(current_problem.lb);

		if (problem_list.empty())
			problem_list.push_front(current_problem);
		else
		{
			list<problem>::iterator it_list = problem_list.begin();
			while (it_list != problem_list.end())
			{
				double current_lb((*it_list).lb);

				if (new_lb >= current_lb)
				{
					problem_list.insert(it_list, current_problem);
					it_list = problem_list.end();
					inserted = 1;
				}
				else
					++it_list;
			}
			if (inserted == 0)
				problem_list.push_back(current_problem);
		}
	}
}

void delete_node()
{
	list<problem>::iterator it_list;
	for (it_list = problem_list.begin(); it_list != problem_list.end();)
	{
		double current_ub((*it_list).ub);
		if (current_ub <= global_lb)
		{
			it_list = problem_list.erase(it_list);
		}
		else
		{
			it_list++;
		}
	}
}

void end()
{
	cout << "global_ub: " << global_ub << endl;
	cout << "global_lb: " << global_lb << endl;
	cout << "B&B node: " << nb_solved_node << endl;
	cout << filename1 << ":	" << filename2 << endl;
	for (int i = 0; i < nb_row; i++)
	{
		cout << i << ":		" << alignment[i] << endl;
	}
	delete[] alignment;
}
